#include "hip/hip_runtime.h"
#define OUT_TILE_DIM 32

__global__ void stencil_kernel(float *in, float* out, unsigned int N){
  int iStart = blockIdx.z * OUT_TILE_DIM;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  float inPrev;
  float inCurr;
  __shared__ float inCurr_s[IN_TILE_DIM][IN_TILE_DIM];
  float inNext;
  if(iStart-1>=0 && iStart-1<N && j>= 0 && j<N && k>=0 && k<N){
    inPrev = in[(iStart-1)*N*N + j*N + k];
  }
  if(iStart>=0 && iStart<N && && j>= 0 && j<N && k>=0 && k<N){
    inCurr = in[iStart*N*N + j*N + k];
    inCurr_s[threadIdx.y][threadIdx.x] = inCurr;
  }
  for(int i = iStart; i< iStart+OUT_TILE_DIM; ++i){
    if(i+1 >=0 && i+1<N && j>= 0 && j<N && k>=0 && k<N ){
      inNext = in[(i+1)*N*N + j*N + k];
    }
    __synchthreads();
    if(i >= 1 && i<N-1 && j>=1 && j<N-1 && k>=1 && k<N-1){
      if(threadIdx.y >= 1 && threadIdx.y < IN_TILE_DIM - 1 && threadIdx.x >= 1 && threadIdx.x < IN_TILE_DIM - 1){
        out[i*N*N + j*N + k] = c0*inCurr + c1*inCurr_s[threadIdx.y][threadIdx.x-1] + c2*inCurr_s[threadIdx.y][threadIdx.x+1]
                                c3*inCurr_s[threadIdx.y-1][threadIdx.x]+c4*inCurr_s[threadIdx.y+1][threadIdx.x]+c5*inPrev+c6*inNext;
      }
    }
    __synchthreads();
    inPrev = inCurr;
    inCurr = inNext;
    inCurr_s[threadIdx.y][threadIdx.x] = inNext;
  }
}
